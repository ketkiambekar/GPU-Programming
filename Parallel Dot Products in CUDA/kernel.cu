#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "dotprod.h"

__global__ void kernel(unsigned int rows, unsigned int cols ,unsigned char *mdata,unsigned char *vdata,float *results){
    /*     unsigned char y;*/
	
        int tid  = threadIdx.x + blockIdx.x * blockDim.x;
    /*int j;*/
	
    /*Coalescent Memory Access*/
		
    /*for(j=0;j<(rows*cols);j++)*/
    if(tid<(rows*cols))
    {
        /*printf("\nThread: %d , mdata:%f",tid,((float)mdata[tid]-48));
        printf("\nThread: %d , vdata:%f",tid,((float)vdata[(tid+1)%cols]-48));
        printf("\nceil: %d.",(tid/cols));*/
        /*printf("\nIndex is : %d. Result is : %d.",(tid%cols),results[(tid/cols)]);*/
        /*printf("\nsize of result %d", sizeof(results[0]));*/
        printf("\nTaking Product of %f and %f:",(float)(mdata[tid]-48),(float)(vdata[(tid%cols)])-48);
        results[(tid/cols)]+=(((float)(mdata[tid]-48))*((float)(vdata[(tid%cols)])-48)));
    }
}

