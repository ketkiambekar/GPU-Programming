#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>
#include "dotprod.h"



int main(int argc ,char* argv[]) 
{

	FILE *fp1;
	FILE *fp2;
	size_t size;
	  
	/* Initialize rows, cols, ncases, ncontrols from the user */
	unsigned int rows=atoi(argv[1]);
    unsigned int cols=atoi(argv[2]);
	int CUDA_DEVICE = atoi(argv[5]);
	int THREADS = atoi(argv[6]);
	printf("rows=%d cols=%d CUDA_DEVICE=%d Threads=%d\n",rows,cols,CUDA_DEVICE,THREADS);

	hipError_t err = hipSetDevice(CUDA_DEVICE);
	if(err != hipSuccess) { printf("Error setting CUDA DEVICE\n"); exit(EXIT_FAILURE); }

	/*Host variable declaration */

	//int THREADS = 32;
	int BLOCKS;
	float* host_results = (float*) malloc(cols * sizeof(float)); 
	struct timeval starttime, endtime;
	clock_t start, end;
	float seconds;
	unsigned int jobs; 
	unsigned long i;


	/*Kernel variable declaration */
	unsigned char *dev_dataM ;
	unsigned char *dev_dataV ;
	float *results;
        char *line = NULL; size_t len = 0;
	char *token, *saveptr;

	start = clock();

	/* Validation to check if the Matrix data file is readable */
	fp1 = fopen(argv[3], "r");
	if (fp1 == NULL) {
    		printf("Cannot Open the File: %s", argv[3]);
		return 0;
    }
    
    /* Validation to check if the Vector data file is readable */
	fp2 = fopen(argv[4], "r");
	if (fp2 == NULL) {
    		printf("Cannot Open the File: %s", argv[4]);
		return 0;
	}

	size = (size_t)((size_t)rows * (size_t)cols);
	printf("Size of the data = %lu\n",size);

	fflush(stdout);

    /*Allocate memory for Matrix*/
    unsigned char *dataM = (unsigned char*)malloc((size_t)size); 

	if(dataM == NULL) {
	        printf("ERROR: Memory for Matrix data not allocated.\n");
    }
    
    /*Allocate memory for Vector*/  /*$$CHECK$$ whether shoukld be row or column*/
    unsigned char *dataV = (unsigned char*)malloc((size_t)cols);

    if(dataV == NULL) {
        printf("ERROR: Memory for Vector data not allocated.\n");
	}

	gettimeofday(&starttime, NULL);

	/* Transfer the Matrix Data from the file to CPU Memory */
	i=0;
	while (getline(&line, &len, fp1) != -1) {
                token = strtok_r(line, " ", &saveptr);
                while(token != NULL){
						dataM[i] = *token;
						printf("\n %d", (int) *token);
                        i++;
                        token = strtok_r(NULL, " ", &saveptr);
                }
		 /* Transfer the Vector Data from the file to CPU Memory */
		}
	
         i=0;
         while (getline(&line, &len, fp2) != -1) {
                     token = strtok_r(line, " ", &saveptr);
                     while(token != NULL){
							 dataV[i] = *token;
							/* printf( *token);*/
							 printf("\n %d", (int) dataV[i] );
                             i++;
                             token = strtok_r(NULL, " ", &saveptr);
                     }
/*
                cur=0; read=-1;
                token = strtok(line, " ");
                while(sscanf(line+cur, "%d%n", &tmp, &read)==1){
                        dataT[i] = (char)(((int)'0')+tmp);
                        cur += read;
                        i++;
                }
*/
  	}
    fclose(fp1);
    fclose(fp2);
        printf("\nData read done.\n");
        fflush(stdout);

        gettimeofday(&endtime, NULL);
        seconds+=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);

        printf("time to read data = %f\n", seconds);

	/* allocate the Memory in the GPU for Matrix data */	   
        gettimeofday(&starttime, NULL);
	err = hipMalloc((unsigned char**) &dev_dataM, (size_t) size * (size_t) sizeof(unsigned char) );
	if(err != hipSuccess) { printf("Error mallocing Matrix data on GPU device\n"); }
        gettimeofday(&endtime, NULL); seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
	printf("time for Matrix cudamalloc=%f\n", seconds);


    
    /* allocate the Memory in the GPU for Vector data */	   
        gettimeofday(&starttime, NULL);
	err = hipMalloc((unsigned char**) &dev_dataV, (size_t) cols * (size_t) sizeof(unsigned char) );
	if(err != hipSuccess) { printf("Error mallocing Vector data on GPU device\n"); }
        gettimeofday(&endtime, NULL); seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
    printf("time for Vector cudamalloc=%f\n", seconds);
    
    /* allocate the Memory in the GPU for Results Vector data */	
        gettimeofday(&starttime, NULL);
	err = hipMalloc((float**) &results, rows * sizeof(float) );
	if(err != hipSuccess) { printf("Error mallocing Vector results on GPU device\n"); }
        gettimeofday(&endtime, NULL); seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
	printf("time for Results cudamalloc=%f\n", seconds);

	/*Copy the Matrix data to GPU */
        gettimeofday(&starttime, NULL);
	err = hipMemcpy(dev_dataM, dataM, (size_t)size * (size_t)sizeof(unsigned char), hipMemcpyHostToDevice);
	if(err != hipSuccess) { printf("Error copying Matrix data to GPU\n"); }
        gettimeofday(&endtime, NULL); seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
    printf("time to copy Matrix Data to GPU=%f\n", seconds);
    
    /*Copy the Vector data to GPU */
        gettimeofday(&starttime, NULL);
	err = hipMemcpy(dev_dataV, dataV, (size_t)cols * (size_t)sizeof(unsigned char), hipMemcpyHostToDevice);
	if(err != hipSuccess) { printf("Error copying Vector data to GPU\n"); }
        gettimeofday(&endtime, NULL); seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
	printf("time to copy Vector Data to GPU=%f\n", seconds);

	jobs = cols;
	BLOCKS = ceil((jobs + THREADS - 1)/THREADS);
	printf("Number of Blocks:%d\n", BLOCKS);
        gettimeofday(&starttime, NULL);

	/*Calling the kernel function */
	kernel<<<BLOCKS,THREADS>>>(rows,cols,dev_dataM,dev_dataV,results);
        gettimeofday(&endtime, NULL); seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
	printf("time for kernel=%f\n", seconds);
		
	/*Copy the results back in host*/
	hipMemcpy(host_results,results,rows * sizeof(float),hipMemcpyDeviceToHost);
	printf("\nResults:\n");
	for(int k = 0; k < jobs; k++) {
		printf("\n %f ", host_results[k]);
	}
	printf("\n");

    hipFree( dev_dataM );
    hipFree( dev_dataV );
	hipFree( results );

	end = clock();
	seconds = (float)(end - start) / CLOCKS_PER_SEC;
	printf("Total time = %f\n", seconds);

	return 0;

}
